#include "cuda/common.cuh"
#include "cuda/encryption.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <cstdint>

bool test_three_b_adapter(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length, const unsigned int number_threads) {
	const auto raw_size = length * sizeof(std::uint64_t);

	std::uint64_t* values_dev = nullptr;
	std::uint64_t* hashes_dev = nullptr;

	hipMalloc((void**)&values_dev, raw_size);
	hipMalloc((void**)&hashes_dev, raw_size);

	hipMemcpy(values_dev, values, raw_size, hipMemcpyHostToDevice);

	auto thread_dim = dim3{ number_threads, 1, 1 };
	auto block_dim = dim3{ 1, 1, 1 };

	flat_hash << <block_dim, thread_dim, HASH_SHARED_MEM >> > (values_dev, hashes_dev, length);

	hipMemcpy(hashes, hashes_dev, raw_size, hipMemcpyDeviceToHost);

	hipFree(values_dev);
	hipFree(hashes_dev);

	hipDeviceSynchronize();

	auto last_error = hipGetLastError();

	return last_error == hipSuccess;
}
