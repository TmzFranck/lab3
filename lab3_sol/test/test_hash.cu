#include "cuda/common.cuh"
#include "cuda/encryption.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <cstdint>

bool test_three_a_adapter(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length, const unsigned int number_threads, const unsigned int number_blocks) {
	const auto raw_size = length * sizeof(std::uint64_t);

	std::uint64_t* values_dev = nullptr;
	std::uint64_t* hashes_dev = nullptr;

	hipMalloc((void**)&values_dev, raw_size);
	hipMalloc((void**)&hashes_dev, raw_size);

	hipMemcpy(values_dev, values, raw_size, hipMemcpyHostToDevice);

	auto thread_dim = dim3{ number_threads, 1, 1 };
	auto block_dim = dim3{ number_blocks, 1, 1 };

	hash << <block_dim, thread_dim, HASH_SHARED_MEM >> > (values_dev, hashes_dev, length);

	hipMemcpy(hashes, hashes_dev, raw_size, hipMemcpyDeviceToHost);

	hipFree(values_dev);
	hipFree(hashes_dev);

	hipDeviceSynchronize();

	auto last_error = hipGetLastError();

	return last_error == hipSuccess;
}

bool test_three_b_adapter(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length, const unsigned int number_threads) {
	const auto raw_size = length * sizeof(std::uint64_t);

	std::uint64_t* values_dev = nullptr;
	std::uint64_t* hashes_dev = nullptr;

	hipMalloc((void**)&values_dev, raw_size);
	hipMalloc((void**)&hashes_dev, raw_size);

	hipMemcpy(values_dev, values, raw_size, hipMemcpyHostToDevice);

	auto thread_dim = dim3{ number_threads, 1, 1 };
	auto block_dim = dim3{ 1, 1, 1 };

	flat_hash << <block_dim, thread_dim, HASH_SHARED_MEM >> > (values_dev, hashes_dev, length);

	hipMemcpy(hashes, hashes_dev, raw_size, hipMemcpyDeviceToHost);

	hipFree(values_dev);
	hipFree(hashes_dev);

	hipDeviceSynchronize();

	auto last_error = hipGetLastError();

	return last_error == hipSuccess;
}

bool test_three_c_adapter(const std::uint64_t* const values, unsigned int* const indices, const unsigned int length, const std::uint64_t searched_hash, const unsigned int number_threads, const unsigned int number_blocks) {
	std::uint64_t* values_dev = nullptr;
	unsigned int* indices_dev = nullptr;
	unsigned int* mem_cell_dev = nullptr;

	hipMalloc((void**)&values_dev, length * sizeof(std::uint64_t));
	hipMalloc((void**)&indices_dev, length * sizeof(unsigned int));
	hipMalloc((void**)&mem_cell_dev, sizeof(unsigned int));

	hipMemcpy(values_dev, values, length * sizeof(std::uint64_t), hipMemcpyHostToDevice);
	hipMemset(indices_dev, 0, length * sizeof(unsigned int));
	hipMemset(mem_cell_dev, 0, sizeof(unsigned int));

	auto thread_dim = dim3{ number_threads, 1, 1 };
	auto block_dim = dim3{ number_blocks, 1, 1 };

	find_hash << <block_dim, thread_dim, FIND_HASH_SHARED_MEM >> > (values_dev, indices_dev, length, searched_hash, mem_cell_dev);

	hipMemcpy(indices, indices_dev, length * sizeof(unsigned int), hipMemcpyDeviceToHost);

	hipFree(values_dev);
	hipFree(indices_dev);
	hipFree(mem_cell_dev);

	hipDeviceSynchronize();

	auto last_error = hipGetLastError();

	return last_error == hipSuccess;
}

bool test_three_d_adapter(std::uint64_t* const hashes, const unsigned int length, const unsigned int number_threads, const unsigned int number_blocks) {
	const auto raw_size = length * sizeof(std::uint64_t);

	std::uint64_t* hashes_dev = nullptr;

	hipMalloc((void**)&hashes_dev, raw_size);

	auto thread_dim = dim3{ number_threads, 1, 1 };
	auto block_dim = dim3{ number_blocks, 1, 1 };

	hash_schemes << <block_dim, thread_dim, HASH_SCHEMES_SHARED_MEM >> > (hashes_dev, length);

	hipMemcpy(hashes, hashes_dev, raw_size, hipMemcpyDeviceToHost);

	hipFree(hashes_dev);

	hipDeviceSynchronize();

	auto last_error = hipGetLastError();

	return last_error == hipSuccess;
}
